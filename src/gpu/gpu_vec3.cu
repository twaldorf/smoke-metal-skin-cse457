#include "hip/hip_runtime.h"
#include "gpu_vec3.cuh"
#include <hiprand/hiprand_kernel.h>

//various vector utility function
__device__ gpu_vec3f gpu_vec3f::random(hiprandState *rand_state)
{
	return { hiprand_uniform(rand_state), hiprand_uniform(rand_state), hiprand_uniform(rand_state)};
}

__device__ gpu_vec3f gpu_vec3f::random(FLOAT min, FLOAT max, hiprandState *rand_state)
{
	return { gpu_random_float(min, max, rand_state), gpu_random_float(min, max, rand_state), gpu_random_float(min, max, rand_state)};
}

__device__ FLOAT gpu_vec3f::length_squared() const {
	return e[0]*e[0] + e[1]*e[1] + e[2]*e[2];
}

__device__ FLOAT gpu_vec3f::length() const
{
	return sqrt(this->length_squared());
}

__device__ gpu_vec3f operator+(const gpu_vec3f &u, const gpu_vec3f &v)
{
	return {u.e[0] + v.e[0], u.e[1] + v.e[1], u.e[2] + v.e[2]};
}

__device__ gpu_vec3f operator-(const gpu_vec3f &u, const gpu_vec3f &v)
{
	return {u.e[0] - v.e[0], u.e[1] - v.e[1], u.e[2] - v.e[2]};
}

__device__ gpu_vec3f operator*(const gpu_vec3f &u, const gpu_vec3f &v)
{
	return {u.e[0] * v.e[0], u.e[1] * v.e[1], u.e[2] * v.e[2]};
}

__device__ gpu_vec3f operator*(FLOAT t, const gpu_vec3f &v)
{
	return {t*v.e[0], t*v.e[1], t*v.e[2]};
}

__device__ gpu_vec3f operator*(const gpu_vec3f &v, FLOAT t)
{
	return t * v;
}

__device__ gpu_vec3f operator/(gpu_vec3f v, FLOAT t)
{
	return (1/t) * v;
}

__device__ FLOAT gpu_dot(const gpu_vec3f &u, const gpu_vec3f &v)
{
	return u.e[0] * v.e[0]
		+ u.e[1] * v.e[1]
		+ u.e[2] * v.e[2];
}

__device__ gpu_vec3f gpu_cross(const gpu_vec3f &u, const gpu_vec3f &v)
{
	return {u.e[1] * v.e[2] - u.e[2] * v.e[1],
			u.e[2] * v.e[0] - u.e[0] * v.e[2],
			u.e[0] * v.e[1] - u.e[1] * v.e[0]};
}

__device__ gpu_vec3f gpu_unit_vector(gpu_vec3f v)
{
	return v / v.length();
}

//diffuse 1
__device__ gpu_vec3f gpu_random_in_unit_sphere(hiprandState *rand_state)
{
	gpu_vec3f p;
	do {
		p = 2.0f*gpu_vec3f(hiprand_uniform(rand_state),hiprand_uniform(rand_state),hiprand_uniform(rand_state)) - gpu_vec3f(1,1,1);
	} while (p.length_squared() >= 1.0f);
	return p;
}

//diffuse 2
__device__ gpu_vec3f gpu_random_unit_vector(hiprandState *rand_state)
{
	return gpu_unit_vector(gpu_random_in_unit_sphere(rand_state));
}

//diffuse 3
__device__ gpu_vec3f gpu_random_in_hemisphere(const gpu_vec3f& normal, hiprandState *rand_state)
{
	gpu_vec3f in_unit_sphere = gpu_random_in_unit_sphere(rand_state);
	if(gpu_dot(in_unit_sphere, normal) > 0.0) // In the same hemisphere as the normal
		return in_unit_sphere;
	else
		return -in_unit_sphere;
}

__device__ bool gpu_vec3f::near_zero() const
{
	// Return true if the vector is close to zero in all dimensions.
	const auto tolerance = 1e-8;
	return (fabs(e[0]) < tolerance) && (fabs(e[1]) < tolerance) && (fabs(e[2]) < tolerance);
}

//calculate direction of a reflected ray
__device__ gpu_vec3f gpu_reflect(const gpu_vec3f& v, const gpu_vec3f& n)
{
	//length of b is v*u then take that and multiply by n to "give it direction"
	return v - 2*gpu_dot(v,n)*n;
}

//calculate a refracted ray's direction
__device__ gpu_vec3f gpu_refract(const gpu_vec3f& uv, const gpu_vec3f& n, FLOAT eta_i_over_eta_t)
{
	FLOAT cos_theta = fmin(gpu_dot(-uv, n), 1.0f);
	gpu_vec3f r_out_perp = eta_i_over_eta_t * (uv + cos_theta*n);
	gpu_vec3f r_out_parallel = -sqrt(fabs(1.0 - r_out_perp.length_squared())) * n;

	return r_out_perp + r_out_parallel;

}

//randomly generate a point in a 2d disk with a radius of 1 (a unit circle)
__device__ gpu_vec3f gpu_random_in_unit_disk(hiprandState *rand_state)
{
	gpu_vec3f p;
	do {
		p = 2.0f*gpu_vec3f(hiprand_uniform(rand_state),hiprand_uniform(rand_state),0) - gpu_vec3f(1,1,0);
	} while (p.length_squared() >= 1.0f);
	return p;
}