#include "hip/hip_runtime.h"
#include <cfloat>
#include <cstdio>
#include "optix_render.cuh"
#include "optix_util.cuh"

//constant medium sphere
template<typename ConstantMediumGeomType>
__device__ void constantMediumSphereBoundsProg(const void* geomData, box3f& primBounds, const int primID)
{
	const ConstantMediumGeomType& self = *(const ConstantMediumGeomType*)geomData;
	const constantMediumSphere& medium = self.prims[primID].constantMediumSphere;
	// Update the bounding box
	primBounds = box3f().extend(medium.sphere.center - medium.sphere.radius).extend(medium.sphere.center + medium.sphere.radius);
}

template<typename ConstantMediumGeomType>
__device__ void constantMediumSphereIntersectProg()
{
	const int primID = optixGetPrimitiveIndex();
	const auto& self = owl::getProgramData<ConstantMediumGeomType>().prims[primID];
	PerRayData& prd = owl::getPRD<PerRayData>();

	const vec3f org = optixGetObjectRayOrigin();
	const vec3f dir = optixGetObjectRayDirection();
	float hit1_t = optixGetRayTmax();
	float hit2_t = optixGetRayTmax();
	const float tmax = optixGetRayTmax();
	const float tmin = optixGetRayTmin();

	const vec3f oc = org - self.constantMediumSphere.sphere.center;

	const float a = dot(dir,dir);
	const float b = dot(oc, dir);
	const float c = dot(oc, oc) - self.constantMediumSphere.sphere.radius * self.constantMediumSphere.sphere.radius;
	const float discriminant = b * b - a * c;

	//	if (!boundary->hit(r, FLT_MIN, FLT_MAX, rec1, rand_state))
	//		return;
	if (discriminant < 0.0f)
	{
		return;
	}

	float root;
	root = (-b + sqrtf(discriminant)) / a;
	if (root < tmax && root > tmin)
		hit1_t = root;
	root = (-b - sqrtf(discriminant)) / a;
	if (root < tmax && root > tmin && root < hit1_t)
		hit1_t = root;
	else if(hit1_t == optixGetRayTmax())
		return;


//	if (!boundary->hit(r, rec1.t+0.0001f, FLT_MAX, rec2, rand_state))
//		return false;
	float temp_tmin = hit1_t+0.0001f;
	root = (-b + sqrtf(discriminant)) / a;
	if (root < tmax && root > temp_tmin)
		hit2_t = root;
	else
	{
		root = (-b - sqrtf(discriminant)) / a;
		if (root < tmax && root > temp_tmin)
			hit2_t = root;
		else
			return;
	}

	if (hit1_t < tmin)
		hit1_t = tmin;
	if (hit2_t > tmax)
		hit2_t = tmax;

	if (hit1_t >= hit2_t)
		return;

	if (hit1_t < 0)
		hit1_t = 0;

	const auto distance_inside_boundary = (hit2_t - hit1_t) * length(dir);
	const auto hit_distance = self.constantMediumSphere.neg_inv_density * log(randomFloat(prd.random));

	if (hit_distance > distance_inside_boundary)
		return;

	hit1_t = hit1_t + hit_distance / length(dir);

	optixReportIntersection(hit1_t, 0);
}

template<typename ConstantMediumGeomType>
__device__ void closestHitConstantMediumSphere()
{
	const int primID = optixGetPrimitiveIndex();
	const auto& self = owl::getProgramData<ConstantMediumGeomType>().prims[primID];

	PerRayData& prd = owl::getPRD<PerRayData>();

	const vec3f org = optixGetWorldRayOrigin();
	const vec3f dir = optixGetWorldRayDirection();
	const float hit_t = optixGetRayTmax();
	const vec3f hit_P = org + hit_t * dir;
	const vec3f N = (hit_P-self.constantMediumSphere.sphere.center);

	// Update the scatter event based on the interaction with the constant medium
	prd.out.scatterEvent = scatter(self.material, hit_P, N, prd) ? rayGotBounced : rayGotCancelled;
}

//SPHERES
template<typename SphereGeomType>
__device__ void sphereBoundsProg(const void *geomData, box3f &primBounds, const int primID)
{
	const SphereGeomType &self = *(const SphereGeomType*)geomData;
	const Sphere sphere = self.prims[primID].sphere;
	primBounds = box3f().extend(sphere.center - sphere.radius).extend(sphere.center + sphere.radius);
}

template<typename SpheresGeomType>
__device__ void sphereIntersectProg()
{
	const int primID = optixGetPrimitiveIndex();
	// printf("isec %i %lx\n",primID,&owl::getProgramData<SpheresGeomType>());
	const auto &self = owl::getProgramData<SpheresGeomType>().prims[primID];

	const vec3f org = optixGetWorldRayOrigin();
	const vec3f dir = optixGetWorldRayDirection();
	float hit_t = optixGetRayTmax();
	const float tmin = optixGetRayTmin();

	const vec3f oc = org - self.sphere.center;

	const float a = dot(dir,dir);
	const float b = dot(oc, dir);
	const float c = dot(oc, oc) - self.sphere.radius * self.sphere.radius;
	const float discriminant = b * b - a * c;

	if (discriminant < 0.f)
		return;

	//locally scope following values
	{
		float temp = (-b - sqrtf(discriminant)) / a;
		if (temp < hit_t && temp > tmin)
			hit_t = temp;
	}
	{
		float temp = (-b + sqrtf(discriminant)) / a;
		if (temp < hit_t && temp > tmin)
			hit_t = temp;
	}

	if (hit_t < optixGetRayTmax())
	{
		optixReportIntersection(hit_t, 0);
	}
}

template<typename SpheresGeomType>
__device__ void closestHitSpheres()
{
	const int primID = optixGetPrimitiveIndex();
	const auto &self = owl::getProgramData<SpheresGeomType>().prims[primID];

	PerRayData &prd = owl::getPRD<PerRayData>();

	const vec3f org = optixGetWorldRayOrigin();
	const vec3f dir = optixGetWorldRayDirection();
	const float hit_t = optixGetRayTmax();
	const vec3f hit_P = org + hit_t * dir;
	const vec3f N = (hit_P-self.sphere.center);

	prd.out.scatterEvent = scatter(self.material, hit_P, N, prd) ? rayGotBounced : rayGotCancelled;
}

//BOXES
template<typename BoxesGeomType>
__device__ void closestHitBoxes()
{
	// printf("chbox\n");
	// return;
	const auto &self = owl::getProgramData<BoxesGeomType>();
	PerRayData &prd = owl::getPRD<PerRayData>();

	// ID of the triangle we've hit:
	const int primID = optixGetPrimitiveIndex();

	// there's 12 tris per box:
	const int materialID = primID / 12;

	const auto &material = self.perBoxMaterial[materialID];

	const vec3i index = self.index[primID];
	const vec3f &A = self.vertex[index.x];
	const vec3f &B = self.vertex[index.y];
	const vec3f &C = self.vertex[index.z];
	const vec3f N = normalize(cross(B-A,C-A));

	const vec3f org = optixGetWorldRayOrigin();
	const vec3f dir = optixGetWorldRayDirection();
	const float hit_t = optixGetRayTmax();
	const vec3f hit_P = org + hit_t * dir;

	prd.out.scatterEvent = scatter(material, hit_P,N, prd) ? rayGotBounced : rayGotCancelled;
}

__device__ vec3f missColor(const Ray &ray)
{
	const vec2i pixelID = owl::getLaunchIndex();

	const vec3f rayDir = normalize(ray.direction);
	const float t = 0.5f*(rayDir.y + 1.0f);
	const vec3f c = (1.0f - t)*vec3f(1.0f, 1.0f, 1.0f) + t * vec3f(0.5f, 0.7f, 1.0f);
	return c;
}

__device__ vec3f tracePath(const RayGenData &self, owl::Ray &ray, PerRayData &prd)
{
	vec3f attenuation = 1.0f;
	printf("WTF\n");
	/* iterative version of recursion, up to depth 50 */
	for (int depth = 0; depth < 50; depth++)
	{
		prd.out.scatterEvent = rayDidntHitAnything;
		owl::traceRay(	/*accel to trace against*/self.world,
						/*the ray to trace*/ ray,
						/*prd*/prd);

		/* ray got 'lost' to the environment - 'light' it with miss
   			shader */
		if (prd.out.scatterEvent == rayDidntHitAnything)
			return attenuation * missColor(ray);
		else if (prd.out.scatterEvent == rayGotCancelled)
			return vec3f(0.0f);

		else
		{
			// ray is still alive, and got properly bounced
			attenuation *= prd.out.attenuation;
			ray = owl::Ray(prd.out.scattered_origin, //origin
							prd.out.scattered_direction, // direction:
							1e-3f, //tmin
							1e10f); //tmax
		}
	}
	// recursion did not terminate - cancel it
	return vec3f(0.0f);
}