
#include <hip/hip_runtime.h>
#ifdef USE_CUDA
#include <cstdlib>
#include <iostream>
#include "cuda.hpp"
#include "sphere.hpp"
#include "material.hpp"
#include "hitable_list.hpp"

void check_cuda(cudaError_t result, char const *const func, const char *const file, int const line)
{
	if (result) {
		std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
				  file << ":" << line << " '" << func << "' \n";
		// Make sure we call CUDA Device Reset before exiting
		cudaDeviceReset();
		exit(99);
	}
}

__global__ void rand_init(curandState *rand_state)
{
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		curand_init(1984, 0, 0, rand_state);
	}
}

__global__ void render_init(int max_x, int max_y, curandState *rand_state)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if((i >= max_x) || (j >= max_y)) return;
	int pixel_index = j*max_x + i;
	// Original: Each thread gets same seed, a different sequence number, no offset
	// curand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
	// BUGFIX, see Issue#2: Each thread gets different seed, same sequence for
	// performance improvement of about 2x!
	curand_init(1984+pixel_index, 0, 0, &rand_state[pixel_index]);
}

__global__ void render(vec3 *fb, int image_width, int image_height, int samples_per_pixel, camera **cam, hitable **world, curandState *rand_state, int max_depth)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	if((i >= image_width) || (j >= image_height))
		return;

	int pixel_index = j*image_width + i;
	curandState local_rand_state = rand_state[pixel_index];
	colour col(0,0,0);

	for(int s=0; s < samples_per_pixel; s++) {
		FLOAT u = FLOAT(i + curand_uniform(&local_rand_state)) / FLOAT(image_width);
		FLOAT v = FLOAT(j + curand_uniform(&local_rand_state)) / FLOAT(image_height);
		ray r = (*cam)->get_ray(u, v, &local_rand_state);
		col += ray_colour(r, world, &local_rand_state, max_depth);
	}

	rand_state[pixel_index] = local_rand_state;

	col /= FLOAT(samples_per_pixel);
	col[0] = sqrt(col[0]);
	col[1] = sqrt(col[1]);
	col[2] = sqrt(col[2]);
	fb[pixel_index] = col;
}

__global__ void create_world(hitable **d_list, hitable **d_world, camera **d_camera, int nx, int ny, curandState *rand_state)
{
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		curandState local_rand_state = *rand_state;
		d_list[0] = new sphere(vec3(0,-1000.0,-1), 1000,
			new lambertian(vec3(0.5, 0.5, 0.5)));
		int i = 1;
		for(int a = -11; a < 11; a++) {
			for(int b = -11; b < 11; b++) {
				FLOAT choose_mat = RND;
				vec3 center(a+RND,0.2,b+RND);
				if(choose_mat < 0.8f) {
					d_list[i++] = new sphere(center, 0.2,
						new lambertian(vec3(RND*RND, RND*RND, RND*RND)));
				}
				else if(choose_mat < 0.95f) {
					d_list[i++] = new sphere(center, 0.2,
						new metal(vec3(0.5f*(1.0f+RND), 0.5f*(1.0f+RND), 0.5f*(1.0f+RND)), 0.5f*RND));
				}
				else {
					d_list[i++] = new sphere(center, 0.2, new dielectric(1.5));
				}
			}
		}
		d_list[i++] = new sphere(vec3(0, 1,0),  1.0, new dielectric(1.5));
		d_list[i++] = new sphere(vec3(-4, 1, 0), 1.0, new lambertian(vec3(0.4, 0.2, 0.1)));
		d_list[i++] = new sphere(vec3(4, 1, 0),  1.0, new metal(vec3(0.7, 0.6, 0.5), 0.0));
		*rand_state = local_rand_state;
		*d_world  = new hitable_list(d_list, 22*22+1+3);

		vec3 lookfrom(13,2,3);
		vec3 lookat(0,0,0);
		FLOAT dist_to_focus = 10.0; (lookfrom-lookat).length();
		FLOAT aperture = 0.1;
		*d_camera   = new camera(lookfrom,
			lookat,
			vec3(0,1,0),
			30.0,
			FLOAT(nx)/FLOAT(ny),
			aperture,
			dist_to_focus);
	}
}

__global__ void free_world(hitable **d_list, hitable **d_world, camera **d_camera)
{
	for(int i=0; i < 22*22+1+3; i++) {
		delete ((sphere *)d_list[i])->mat_ptr;
		delete d_list[i];
	}
	delete *d_world;
	delete *d_camera;
}

#endif